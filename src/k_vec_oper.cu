#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "matrix.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define ERRPRINT(text) \
err = hipGetLastError();\
if(err != hipSuccess)\
  printf("%s: %s\n", text, hipGetErrorString(err));



time_s Operation(const matrix& arr1, const matrix& arr2, matrix& out, const Oper& op) {
  time_s time;
  hipError_t err;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;
  uint32_t *d_size;
  hipMalloc((void**)&d_arr1, arr1.size*arr1.size*sizeof(arr_t));
  hipMalloc((void**)&d_arr2, arr2.size*arr2.size*sizeof(arr_t));
  hipMalloc((void**)&d_out, out.size*out.size*sizeof(arr_t));
  hipMalloc((void**)&d_size, sizeof(uint32_t));

  float *h_arr1 = (arr_t*)malloc(arr1.size*arr1.size*sizeof(arr_t)),
        *h_arr2 = (arr_t*)malloc(arr2.size*arr2.size*sizeof(arr_t)),
        *flat_out = (arr_t*)malloc(out.size*out.size*sizeof(arr_t));
  hipStream_t stream;
  hipStreamCreate(&stream);

  ERRPRINT("Malloc");

  hipHostRegister(h_arr1, arr1.size*arr1.size*sizeof(arr_t), hipHostRegisterDefault);
  hipHostRegister(h_arr2, arr2.size*arr2.size*sizeof(arr_t), hipHostRegisterDefault);
  CUDATIME(({
    arr1.flat(h_arr1);
    arr2.flat(h_arr2);

    hipMemcpyAsync(d_arr1, h_arr1, arr1.size*arr1.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_arr2, h_arr2, arr2.size*arr2.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemcpy(d_size, &out.size, sizeof(uint32_t), hipMemcpyHostToDevice); 
  }), time.memcpy, start, end);
  hipHostUnregister(h_arr1);
  hipHostUnregister(h_arr2);
  free(h_arr1);
  free(h_arr2);

  ERRPRINT("Memcpy");

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  void (*func)(arr_t*, arr_t*, arr_t*, const uint32_t&) = nullptr;

  CUDATIME(({
    switch(op) {
      case opadd : func = KAdd; break;
      case opsub : func = KSub; break;
      case opmul : func = KMul; break;
      case opdiv : func = KDiv; break;
      default : break;
    }
    if(func)
      func<<<blocks, threads>>>(d_arr1, d_arr2, d_out, *d_size * *d_size);
  }), time.run, start, end);

  ERRPRINT("Running");

  CUDATIME(({
    hipHostRegister(flat_out, out.size*out.size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(flat_out, d_out, out.size*out.size*sizeof(arr_t), hipMemcpyDeviceToHost, stream);

    hipHostUnregister(flat_out);
    out.deflat(flat_out);
  }), time.memret, start, end);

  ERRPRINT("Return");

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);
  hipFree(d_size);

  ERRPRINT("Free");

  return time;
}



__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
