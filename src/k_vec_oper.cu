#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "matrix.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(const matrix& arr1, const matrix& arr2, matrix& out, const Oper& op) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;\
  hipMalloc((void**)&d_arr1, arr1.size*arr1.size*sizeof(arr_t));\
  hipMalloc((void**)&d_arr2, arr2.size*arr2.size*sizeof(arr_t));\
  hipMalloc((void**)&d_out, out.size*out.size*sizeof(arr_t));\

  arr_t* h_arr1 = arr1.flat();
  arr_t* h_arr2 = arr2.flat();
  arr_t* temp_out;


  CUDATIME(({
    hipMemcpy(d_arr1, h_arr1, arr1.size*arr1.size*sizeof(arr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, h_arr2, arr2.size*arr2.size*sizeof(arr_t), hipMemcpyHostToDevice);
  }), time.memcpy, start, end);

  free(h_arr1);
  free(h_arr2);

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  CUDATIME(({
    switch(op) {
      case opadd : KAdd<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opsub : KSub<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opmul : KMul<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opdiv : KDiv<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      default : break;
    } 
  }), time.run, start, end);

  CUDATIME(({
    hipMemcpy(temp_out, d_out, size, hipMemcpyDeviceToHost);
    out.deflat(temp_out);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);

  return time;
}



__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
