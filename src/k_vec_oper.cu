#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "matrix.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(const matrix& arr1, const matrix& arr2, matrix& out, const Oper& op) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;\
  hipMalloc((void**)&d_arr1, arr1.size*arr1.size*sizeof(arr_t));\
  hipMalloc((void**)&d_arr2, arr2.size*arr2.size*sizeof(arr_t));\
  hipMalloc((void**)&d_out, out.size*out.size*sizeof(arr_t));\

  hipStream_t stream;
  hipStreamCreate(&stream); 


  CUDATIME(({
    for(uint32_t i=0; i<out.size; ++i) {
      hipHostRegister(arr1.data[i], arr1.size*sizeof(arr_t), hipHostRegisterDefault);
      hipHostRegister(arr2.data[i], arr2.size*sizeof(arr_t), hipHostRegisterDefault);

      hipMemcpyAsync(&d_arr1[i*out.size], arr1.data[i], arr1.size*arr1.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
      hipMemcpyAsync(&d_arr2[i*out.size], arr2.data[i], arr2.size*arr2.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);

      hipHostUnregister(arr1.data[i]);
      hipHostUnregister(arr2.data[i]);
    }
  }), time.memcpy, start, end);

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  CUDATIME(({
    switch(op) {
      case opadd : KAdd<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opsub : KSub<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opmul : KMul<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opdiv : KDiv<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      default : break;
    } 
  }), time.run, start, end);

  CUDATIME(({
    for(uint32_t i=0; i<out.size; ++i) {
      hipHostRegister(out.data[i], out.size*sizeof(arr_t), hipHostRegisterDefault);

      hipMemcpyAsync(out.data[i], &d_out[i*out.size], out.size, hipMemcpyDeviceToHost, stream);
      
      hipHostUnregister(out.data[i]);
    }
  }), time.memret, start, end);

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);

  return time;
}



__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
