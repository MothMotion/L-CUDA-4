#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



extern "C" time_s Operation(arr_t** arr1, arr_t** arr2, arr_t** out, const uint32_t size, const enum Oper op) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;\
  hipMalloc((void**)&d_arr1, size*sizeof(arr_t));\
  hipMalloc((void**)&d_arr2, size*sizeof(arr_t));\
  hipMalloc((void**)&d_out, size*sizeof(arr_t));\


  CUDATIME(({
    hipMemcpy(d_arr1, arr1, size*sizeof(arr_t), hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, arr2, size*sizeof(arr_t), hipMemcpyHostToDevice); 
  }), time.memcpy, start, end);

  CUDATIME(({
    switch(op) {
      case opadd : KAdd<<<KBLOCKS,KTHREADS>>>(d_arr1, d_arr2, d_out, size); break;
      case opsub : KSub<<<KBLOCKS,KTHREADS>>>(d_arr1, d_arr2, d_out, size); break;
      case opmul : KMul<<<KBLOCKS,KTHREADS>>>(d_arr1, d_arr2, d_out, size); break;
      case opdiv : KDiv<<<KBLOCKS,KTHREADS>>>(d_arr1, d_arr2, d_out, size); break;
    } 
  }), time.run, start, end);

  CUDATIME(({
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
  }), time.memret, start, end);

  time.total = time.memcpy + time.run + time.memret;

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);

  return time;
}

__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, uint32_t size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, uint32_t size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, uint32_t size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, uint32_t size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
