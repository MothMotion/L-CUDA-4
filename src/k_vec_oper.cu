#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "matrix.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(const matrix& arr1, const matrix& arr2, matrix& out, const Oper& op) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t *d_arr1, *d_arr2, *d_out;\
  hipMalloc((void**)&d_arr1, arr1.size*arr1.size*sizeof(arr_t));\
  hipMalloc((void**)&d_arr2, arr2.size*arr2.size*sizeof(arr_t));\
  hipMalloc((void**)&d_out, out.size*out.size*sizeof(arr_t));\

  float *h_arr1, *h_arr2, *flat_out;
  hipStream_t stream;
  hipStreamCreate(&stream); 


  CUDATIME(({
    h_arr1 = arr1.flat();
    h_arr2 = arr2.flat();
    flat_out = (arr_t*)malloc(out.size*out.size*sizeof(arr_t));

    hipHostRegister(h_arr1, arr1.size*arr1.size, hipHostRegisterDefault);
    hipHostRegister(h_arr2, arr2.size*arr2.size, hipHostRegisterDefault);
    hipHostRegister(flat_out, out.size*out.size, hipHostRegisterDefault);

    hipMemcpyAsync(d_arr1, h_arr1, arr1.size*arr1.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_arr2, h_arr2, arr2.size*arr2.size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
  }), time.memcpy, start, end);

  hipHostUnregister(h_arr1);
  free(h_arr1);
  hipHostUnregister(h_arr2);
  free(h_arr2);

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  CUDATIME(({
    switch(op) {
      case opadd : KAdd<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opsub : KSub<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opmul : KMul<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      case opdiv : KDiv<<<blocks,threads>>>(d_arr1, d_arr2, d_out, out.size*out.size); break;
      default : break;
    } 
  }), time.run, start, end);

  CUDATIME(({
    hipMemcpyAsync(flat_out, d_out, size, hipMemcpyDeviceToHost, stream);
    hipHostUnregister(flat_out);
    out.deflat(flat_out);
  }), time.memret, start, end);

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr1);
  hipFree(d_arr2);
  hipFree(d_out);

  return time;
}



__global__ void KAdd(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] + arr2[idx];
}

__global__ void KSub(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] - arr2[idx];
}

__global__ void KMul(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size)
    out[idx] = arr1[idx] * arr2[idx];
}

__global__ void KDiv(arr_t* arr1, arr_t* arr2, arr_t* out, const uint32_t& size) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < size && arr2[idx])
    out[idx] = arr1[idx] / arr2[idx];
}

#endif
